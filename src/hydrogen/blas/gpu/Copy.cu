#include "hip/hip_runtime.h"
#include <hydrogen/blas/gpu/Copy.hpp>

#include <El/hydrogen_config.h>
#include <hydrogen/device/gpu/CUDA.hpp>
#include <hip/hip_runtime.h>

namespace
{

template <typename SrcT, typename DestT, typename SizeT>
__global__ void copy_1d_kernel(
    SizeT num_entries,
    SrcT const* __restrict__ src, SizeT src_stride,
    DestT* __restrict__ dest, SizeT dest_stride)
{
    SizeT const idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_entries)
        dest[idx*dest_stride] = src[idx*src_stride];
}

// FIXME (trb): It's not clear to me that this is the "right" kernel
// for row_stride > 1. Cache performance likely gets trashed such that
// "row_stride" times as many cache misses occur.
template <int TILE_SIZE, int BLK_COLS,
          typename SrcT, typename DestT, typename SizeT>
__global__ void copy_2d_kernel(
    SizeT m, SizeT n,
    SrcT const* __restrict__ src, SizeT src_row_stride, SizeT src_col_stride,
    DestT* __restrict__ dest, SizeT dest_row_stride, SizeT dest_col_stride)
{
    __shared__ SrcT tile[TILE_SIZE][TILE_SIZE+1];

    SizeT const start_row = blockIdx.x * TILE_SIZE + threadIdx.x;
    SizeT const start_col = blockIdx.y * TILE_SIZE + threadIdx.y;

    src += start_row*src_row_stride + start_col*src_col_stride;
    dest += start_row*dest_row_stride + start_col*dest_col_stride;
    if (start_row < m && start_col < n)
    {
        if (start_col + TILE_SIZE < n)
        {
            // Load the data
            #pragma unroll
            for (int ii = 0; ii < TILE_SIZE; ii += BLK_COLS)
                tile[threadIdx.y+ii][threadIdx.x] = src[ii*src_col_stride];

            // Store the data
            #pragma unroll
            for (int ii = 0; ii < TILE_SIZE; ii += BLK_COLS)
                dest[ii*dest_col_stride] = tile[threadIdx.y+ii][threadIdx.x];
        }
        else
        {
            // Load the data
            for (int ii = 0;
                 ii < TILE_SIZE && start_col + ii < n; ii += BLK_COLS)
            {
                tile[threadIdx.y+ii][threadIdx.x] = src[ii*src_col_stride];
            }

            // Store the data
            for (int ii = 0;
                 ii < TILE_SIZE && start_col + ii < n; ii += BLK_COLS)
            {
                dest[ii*dest_col_stride] = tile[threadIdx.y+ii][threadIdx.x];
            }
        }
    }
}

}// namespace <anon>

namespace hydrogen
{

template <typename SrcT, typename DestT, typename SizeT, typename, typename>
void Copy_GPU_impl(
    SizeT num_entries,
    SrcT const* src, SizeT src_stride,
    DestT * dest, SizeT dest_stride,
    hipStream_t stream)
{
    if (num_entries <= TypeTraits<SizeT>::Zero())
        return;

#ifdef HYDROGEN_DO_BOUNDS_CHECKING
    // The kernel parameters are __restrict__-ed. This helps ensure
    // that's not a lie.
    void const* max_src = src + src_stride*num_entries;
    void const* max_dest = dest + dest_stride*num_entries;
    if ((dest < max_src) && (src < max_dest))
        throw std::logic_error(
            "Overlapping memory regions are not allowed.");
#endif // HYDROGEN_DO_BOUNDS_CHECKING

    constexpr size_t threads_per_block = 128;
    auto blocks = (num_entries + threads_per_block - 1)/ threads_per_block;
    void* args[] = { &num_entries, &src, &src_stride, &dest, &dest_stride };

    H_CHECK_CUDA(
        hipLaunchKernel(
            (void const*)&copy_1d_kernel<SrcT,DestT,SizeT>,
            blocks, threads_per_block,
            args, 0, stream));
}

template <typename SrcT, typename DestT, typename SizeT, typename, typename>
void Copy_GPU_impl(
    SizeT num_rows, SizeT num_cols,
    SrcT const* src, SizeT src_row_stride, SizeT src_col_stride,
    DestT* dest, SizeT dest_row_stride, SizeT dest_col_stride,
    hipStream_t stream)
{
  if (num_rows == 0 || num_cols == 0)
    return;

#ifdef HYDROGEN_DO_BOUNDS_CHECKING
    // The kernel parameters are __restrict__-ed. This helps ensure
    // that's not a lie.
    void const* max_src = src + src_col_stride*num_cols;
    void const* max_dest = dest + dest_col_stride*num_cols;
    if ((dest < max_src) && (src < max_dest))
        throw std::logic_error(
            "Overlapping memory regions are not allowed.");
#endif // HYDROGEN_DO_BOUNDS_CHECKING

    constexpr int TILE_SIZE = 32;
    constexpr int BLK_COLS = 8;

    dim3 blks((num_rows + TILE_SIZE - 1)/TILE_SIZE,
              (num_cols + TILE_SIZE - 1)/TILE_SIZE, 1);
    dim3 thds(TILE_SIZE, BLK_COLS, 1);
    void* args[] = { &num_rows, &num_cols,
                     &src, &src_row_stride, &src_col_stride,
                     &dest, &dest_row_stride, &dest_col_stride };

    H_CHECK_CUDA(
        hipLaunchKernel(
            (void const*)&copy_2d_kernel<TILE_SIZE,BLK_COLS,SrcT,DestT,SizeT>,
            blks, thds, args, 0, stream));
}

#define ETI(SourceType, DestType, SizeType)             \
    template void Copy_GPU_impl(                        \
        SizeType, SourceType const*, SizeType,          \
        DestType*, SizeType, hipStream_t);             \
    template void Copy_GPU_impl(                        \
        SizeType, SizeType,                             \
        SourceType const*, SizeType, SizeType,          \
        DestType*, SizeType, SizeType, hipStream_t)

ETI(float, float, int);
ETI(float, float, long);
ETI(float, float, long long);
ETI(float, float, unsigned);
ETI(float, float, size_t);

ETI(float, double, int);
ETI(float, double, long);
ETI(float, double, long long);
ETI(float, double, unsigned);
ETI(float, double, size_t);

ETI(double, float, int);
ETI(double, float, long);
ETI(double, float, long long);
ETI(double, float, unsigned);
ETI(double, float, size_t);

ETI(double, double, int);
ETI(double, double, long);
ETI(double, double, long long);
ETI(double, double, unsigned);
ETI(double, double, size_t);

#ifdef HYDROGEN_GPU_USE_FP16
ETI(gpu_half_type, gpu_half_type, int);
ETI(gpu_half_type, gpu_half_type, long);
ETI(gpu_half_type, gpu_half_type, long long);
ETI(gpu_half_type, gpu_half_type, unsigned);
ETI(gpu_half_type, gpu_half_type, size_t);

ETI(gpu_half_type, float, int);
ETI(gpu_half_type, float, long);
ETI(gpu_half_type, float, long long);
ETI(gpu_half_type, float, unsigned);
ETI(gpu_half_type, float, size_t);

ETI(float, gpu_half_type, int);
ETI(float, gpu_half_type, long);
ETI(float, gpu_half_type, long long);
ETI(float, gpu_half_type, unsigned);
ETI(float, gpu_half_type, size_t);

ETI(gpu_half_type, double, int);
ETI(gpu_half_type, double, long);
ETI(gpu_half_type, double, long long);
ETI(gpu_half_type, double, unsigned);
ETI(gpu_half_type, double, size_t);

ETI(double, gpu_half_type, int);
ETI(double, gpu_half_type, long);
ETI(double, gpu_half_type, long long);
ETI(double, gpu_half_type, unsigned);
ETI(double, gpu_half_type, size_t);
#endif

#ifdef HYDROGEN_HAVE_SHORT
ETI(short, short, int);
ETI(short, short, long);
ETI(short, short, long long);
ETI(short, short, unsigned);
ETI(short, short, size_t);

ETI(short, float, int);
ETI(short, float, long);
ETI(short, float, long long);
ETI(short, float, unsigned);
ETI(short, float, size_t);

ETI(float, short, int);
ETI(float, short, long);
ETI(float, short, long long);
ETI(float, short, unsigned);
ETI(float, short, size_t);

ETI(short, double, int);
ETI(short, double, long);
ETI(short, double, long long);
ETI(short, double, unsigned);
ETI(short, double, size_t);

ETI(double, short, int);
ETI(double, short, long);
ETI(double, short, long long);
ETI(double, short, unsigned);
ETI(double, short, size_t);
#endif

}// namespace hydrogen
