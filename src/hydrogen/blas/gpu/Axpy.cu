#include "hip/hip_runtime.h"
#include <hydrogen/blas/gpu/Axpy.hpp>

#include <El/hydrogen_config.h>
#include <hydrogen/device/gpu/CUDA.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

namespace
{

// NOTE: B has dimension m x n.
template <int TILE_SIZE, int BLK_COLS, typename T, typename SizeT>
__global__ void axpy_2d_transpose_tiled_kernel(
    SizeT m, SizeT n, T alpha, T const* A, SizeT lda, T* B, SizeT ldb)
{

    // All the fun of a transpose meets the awesomeness of Axpy. :D
    //
    // remember: B is m x n, A is n x m
    cg::thread_block cta = cg::this_thread_block();
    __shared__ T tile[TILE_SIZE][TILE_SIZE+1];

    auto const row_start_A = blockIdx.y * TILE_SIZE + threadIdx.x;
    auto const col_start_A = blockIdx.x * TILE_SIZE + threadIdx.y;

    A += row_start_A + col_start_A * lda;

    auto const row_start_B = blockIdx.x * TILE_SIZE + threadIdx.x;
    auto const col_start_B = blockIdx.y * TILE_SIZE + threadIdx.y;

    B += row_start_B + col_start_B * ldb;

    // If I am a valid row in A, I need to load things
    if (row_start_A < n)
    {
        if (col_start_A + TILE_SIZE <= m)
        {
            #pragma unroll
            for (int ii = 0; ii < TILE_SIZE; ii += BLK_COLS)
                tile[threadIdx.y+ii][threadIdx.x] = alpha * A[ii*lda];
        }
        else
        {
            for (int ii = 0; ii < TILE_SIZE && col_start_A + ii < m; ii += BLK_COLS)
                tile[threadIdx.y+ii][threadIdx.x] = alpha * A[ii*lda];
         }
    }

    cg::sync(cta);

    // If I am a valid row in B, I need to store things
    if (row_start_B < m)
    {
        if (col_start_B + TILE_SIZE <= n)
        {
            #pragma unroll
            for (int ii = 0; ii < TILE_SIZE; ii += BLK_COLS)
                B[ii*ldb] += tile[threadIdx.x][threadIdx.y+ii];
        }
        else
        {
            for (int ii = 0; ii < TILE_SIZE && col_start_B + ii < n; ii += BLK_COLS)
                B[ii*ldb] += tile[threadIdx.x][threadIdx.y+ii];
        }
    }
}

template <int TILE_SIZE, int BLK_COLS, typename T, typename SizeT>
__global__ void axpy_2d_tiled_kernel(
    SizeT m, SizeT n, T alpha,
    T const* A, SizeT row_stride_A, SizeT col_stride_A,
    T* B, SizeT row_stride_B, SizeT col_stride_B)
{
    auto row_start = blockIdx.x * TILE_SIZE + threadIdx.x;
    auto col_start = blockIdx.y * TILE_SIZE + threadIdx.y;

    auto idx_in = row_start*row_stride_A + col_start*col_stride_A;
    auto idx_out = row_start*row_stride_B + col_start*col_stride_B;

    if (row_start < m)
    {
        A += idx_in;
        B += idx_out;
        if (col_start + TILE_SIZE <= n)
        {
            #pragma unroll
            for (int ii = 0; ii < TILE_SIZE; ii += BLK_COLS)
                B[ii*col_stride_B] += alpha * A[ii*col_stride_A];
        }
        else
        {
            for (int ii = 0; ii < TILE_SIZE && col_start + ii < n; ii += BLK_COLS)
                B[ii*col_stride_B] += alpha * A[ii*col_stride_A];
        }
    }
}

}// namespace <anon>

namespace hydrogen
{

template <typename T, typename SizeT, typename>
void Axpy_GPU_impl(
    SizeT height, SizeT width,
    T alpha,
    T const* X, SizeT colStrideX, SizeT rowStrideX,
    T* Y, SizeT colStrideY, SizeT rowStrideY,
    hipStream_t stream)
{
    if (height == TypeTraits<SizeT>::Zero()
        || width == TypeTraits<SizeT>::Zero())
    {
        return;
    }

    constexpr int TILE_SIZE = 32;
    constexpr int BLK_COLS = 8;

    // Short-circuit
    if (height <= 0 || width <= 0)
        return;

    dim3 blks((height + TILE_SIZE - 1) / TILE_SIZE,
              (width + TILE_SIZE - 1) / TILE_SIZE, 1);
    dim3 thds(TILE_SIZE, BLK_COLS, 1);
    void* args[] = {&height, &width, &alpha,
                    &X, &colStrideX, &rowStrideX,
                    &Y, &colStrideY, &rowStrideY};

    H_CHECK_CUDA(
        hipLaunchKernel(
            (void const*)&axpy_2d_tiled_kernel<TILE_SIZE, BLK_COLS, T, SizeT>,
            blks, thds, args, 0, stream));
}

template <typename T, typename SizeT, typename>
void Axpy_GPU_impl(
    TransposeMode transpA,
    SizeT height, SizeT width,
    T alpha,
    T const* A, SizeT lda,
    T* B, SizeT ldb,
    hipStream_t stream)
{
    // Short-circuit
    if (height <= TypeTraits<SizeT>::Zero()
        || width <= TypeTraits<SizeT>::Zero())
    {
        return;
    }

    if (transpA == TransposeMode::NORMAL)
        return Axpy_GPU_impl(
            height, width, alpha,
            A, TypeTraits<SizeT>::One(), lda,
            B, TypeTraits<SizeT>::One(), ldb, stream);

    constexpr int TILE_SIZE = 32;
    constexpr int BLK_COLS = 8;

    dim3 blks((height + TILE_SIZE - 1) / TILE_SIZE,
              (width + TILE_SIZE - 1) / TILE_SIZE, 1);
    dim3 thds(TILE_SIZE, BLK_COLS, 1);
    void* args[] = {&height, &width, &alpha, &A, &lda, &B, &ldb};

    H_CHECK_CUDA(
        hipLaunchKernel(
            (void const*)&axpy_2d_transpose_tiled_kernel
            <TILE_SIZE, BLK_COLS, T, SizeT>,
            blks, thds, args, 0, stream));
}

#define ETI(ScalarT, SizeT)                             \
    template void Axpy_GPU_impl(                        \
        SizeT, SizeT, ScalarT,                          \
        ScalarT const*, SizeT, SizeT,                   \
        ScalarT*, SizeT, SizeT, hipStream_t);          \
    template void Axpy_GPU_impl(                        \
        TransposeMode, SizeT, SizeT, ScalarT,           \
        ScalarT const*, SizeT,                          \
        ScalarT*, SizeT, hipStream_t)


#ifdef HYDROGEN_GPU_USE_FP16
ETI(gpu_half_type, int);
ETI(gpu_half_type, long);
ETI(gpu_half_type, long long);
ETI(gpu_half_type, unsigned);
ETI(gpu_half_type, size_t);
#endif

ETI(float, int);
ETI(float, long);
ETI(float, long long);
ETI(float, unsigned);
ETI(float, size_t);

ETI(double, int);
ETI(double, long);
ETI(double, long long);
ETI(double, unsigned);
ETI(double, size_t);

#ifdef HYDROGEN_HAVE_SHORT
ETI(short, int);
ETI(short, long);
ETI(short, long long);
ETI(short, unsigned);
ETI(short, size_t);
#endif

}// namespace hydrogen
